// Copyright (c) 2009-2018 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeSphere.h"
#include "ShapeConvexPolygon.h"
#include "ShapePolyhedron.h"
#include "ShapeSpheropolyhedron.h"
#include "ShapeSpheropolyhedron.h"
#include "ShapeSpheropolygon.h"
#include "ShapeSimplePolygon.h"
#include "ShapeEllipsoid.h"
#include "ShapeFacetedSphere.h"
#include "ShapeSphinx.h"
#include "ShapeUnion.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeUnion<ShapeSpheropolyhedron>
template hipError_t gpu_hpmc_free_volume<ShapeUnion<ShapeSpheropolyhedron> >(const hpmc_free_volume_args_t &args,
                                                  const typename ShapeUnion<ShapeSpheropolyhedron> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeUnion<ShapeSpheropolyhedron> >(const hpmc_args_t& args,
                                                  const typename ShapeUnion<ShapeSpheropolyhedron> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeUnion<ShapeSpheropolyhedron> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeSpheropolyhedron> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeUnion<ShapeSpheropolyhedron> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeSpheropolyhedron> ::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeUnion<ShapeSpheropolyhedron> >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeUnion<ShapeSpheropolyhedron> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeUnion<ShapeSpheropolyhedron> >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeUnion<ShapeSpheropolyhedron> ::param_type *d_params);
}; // end namespace detail

} // end namespace hpmc
